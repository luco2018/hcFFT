#include <stdio.h>
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 128 
#define NY 128 
#define NZ 128 
#define BATCH 1

int main()
{
	hipfftHandle plan;
	int Csize = ((NX/2) + 1) * NY * NZ;
	hipfftComplex *input = (hipfftComplex*)calloc(Csize, sizeof(hipfftComplex));
	hipfftComplex *output = (hipfftComplex*)calloc(Csize, sizeof(hipfftComplex));
	hipfftComplex *idata;
	hipfftComplex *odata;

	int seed = 123456789;
	srand(seed);

	for(int i=0; i<Csize; i++)
	{
		input[i].x = rand();
		input[i].y = rand();
	}

	hipMalloc((void**)&idata, sizeof(hipfftComplex)*Csize*BATCH);
	hipMemcpy(idata, input, sizeof(hipfftComplex)*Csize*BATCH, hipMemcpyHostToDevice);
	hipMalloc((void**)&odata, sizeof(hipfftComplex)*Csize*BATCH);
	hipMemcpy(odata, output, sizeof(hipfftComplex)*Csize*BATCH, hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to allocate\n"); 
		return 0;	
	} 

	/* Create a 3D FFT plan. */ 
	if (hipfftPlan3d(&plan, NX, NY, NZ, HIPFFT_C2C) != HIPFFT_SUCCESS) 
	{ 
		fprintf(stderr, "CUFFT error: Plan creation failed"); 
		return 0;	
	}	

	/* Use the CUFFT plan to transform the signal in place. */ 
	if (hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex*)odata, HIPFFT_FORWARD) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed"); 
		return 0;	
	}
	if (hipfftExecC2C(plan, (hipfftComplex*)idata, (hipfftComplex*)odata, HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
	{ 
		fprintf(stderr, "CUFFT error: ExecC2C Inverse failed"); 
		return 0;	
	} 
	if (hipDeviceSynchronize() != hipSuccess)
	{ 
		fprintf(stderr, "Cuda error: Failed to synchronize\n"); 
		return 0;	
	}	
	
	hipfftDestroy(plan); 

	free(input);
	free(output);

	hipFree(idata);
	hipFree(odata);

	return 0;
}
